
#include <hip/hip_runtime.h>
/**
 * Copyright (c) 2010 Open Information Security Foundation.
 *
 * \author Anoop Saldanha <poonaatsoc@gmail.com>
 *
 * \file The Cuda kernel for MPM B2G.
 *
 * \todo This is a basic version of the kernel.  Modify it to support multiple
 *       blocks of threads.  Make use of shared memory/texture memory.
 */

#define B2G_CUDA_Q 2
#define CUDA_THREADS 16
#define B2G_CUDA_HASHSHIFT 4
#define B2G_CUDA_TYPE unsigned int
#define B2G_CUDA_HASH16(a, b) (((a) << B2G_CUDA_HASHSHIFT) | (b))
#define u8_tolower(c) g_u8_lowercasetable[(c)]

extern "C"
__global__ void B2gCudaSearchBNDMq(unsigned int *offsets,
                                   unsigned int *B2G,
                                   unsigned char *g_u8_lowercasetable,
                                   unsigned char *buf,
                                   unsigned short arg_buflen,
                                   unsigned int m)
{
    unsigned int pos = m - B2G_CUDA_Q + 1;
    B2G_CUDA_TYPE d;
    unsigned short h;
    unsigned int j;
    unsigned int first;
    unsigned int tid = threadIdx.x;
    unsigned short tid_chunk = arg_buflen / CUDA_THREADS;
    unsigned short jump;
    unsigned short buflen;

    if (tid_chunk < m)
        tid_chunk = m;

    jump = tid_chunk * tid;
    if ((jump + tid_chunk) > arg_buflen)
        return;

    buflen = tid_chunk * 2 - 1;
    if ((tid == CUDA_THREADS - 1) || ((jump + buflen) > arg_buflen)) {
        buflen = arg_buflen - jump;
    }

    j = 0;
    while (j < buflen) {
        offsets[jump + j] = 0;
        j++;
    }

    while (pos <= (buflen - B2G_CUDA_Q + 1)) {
        h = B2G_CUDA_HASH16(u8_tolower(buf[jump + pos - 1]), u8_tolower(buf[jump + pos]));
        d = B2G[h];

        if (d != 0) {
            j = pos;
            first = pos - (m - B2G_CUDA_Q + 1);

            do {
                j = j - 1;
                if (d >= (1 << (m - 1))) {
                    if (j > first) {
                        pos = j;
                    } else {
                        offsets[j + jump] = 1;
                    }
                }

                if (j == 0)
                    break;

                h = B2G_CUDA_HASH16(u8_tolower(buf[jump + j - 1]), u8_tolower(buf[jump + j]));
                d = (d << 1) & B2G[h];
            } while (d != 0);
        }
        pos = pos + m - B2G_CUDA_Q + 1;
    }

    return;
}
